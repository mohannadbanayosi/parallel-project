#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include <stdio.h>
#include <assert.h>

#ifndef __HIPCC__
#define __HIPCC__
#endif

inline __global__ void MatrixMulKernelTiled8x8(float* Md, float* Nd, float* Pd, int Width);
inline __global__ void MatrixMulKernelTiled16x16(float* Md, float* Nd, float* Pd, int Width);
inline __global__ void MatrixMulKernelTiledUnrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling);
inline __global__ void MatrixMulKernelTiled8x8prefetch(float* Md, float* Nd, float* Pd, int Width);
inline __global__ void MatrixMulKernelTiled8x8gran1x2(float* Md, float* Nd, float* Pd, int Width);
inline __global__ void MatrixMulKernelTiled8x8gran1x4(float* Md, float* Nd, float* Pd, int Width);

inline void MatrixMulOnDevice1(float* M, float* N, float* P, int Width, float timing) {   
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);    
	for (int i = 0; i < Width; ++i) {
        for (int j = 0; j < Width; ++j) {
            double sum = 0;
            for (int k = 0; k < Width; ++k) {
                double a = M[i * Width + k];
                double b = N[k * Width + j];
                sum += a * b;
            }
            P[i*Width+j]=sum;
        }
    }
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timing, start, stop); // that's our time!
	hipEventDestroy(start);
	hipEventDestroy(stop);
}


inline void MatrixMulOnDevice2(float* M, float* N, float* P, int Width, float timing) {   
	int size = Width * Width * sizeof(float);    
	float *Md, *Nd, *Pd;   
	//Allocate and Load M, N to device memory    
	hipMalloc(&Md, size);   
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);   
	hipMalloc(&Nd, size);   
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);   
	//Allocate P on the device   
	hipMalloc(&Pd, size);
	// Kernel invocation code � to be shown later    
	// Setup the execution configuration    
	dim3 dimGrid2(Width/8, Width/8);   
	dim3 dimBlock2(8, 8);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);
	// Do something on GPU
    // Launch the device computation threads!    
	MatrixMulKernelTiled8x8<<< dimGrid2, dimBlock2 >>>(Md, Nd, Pd, Width);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timing, start, stop); // that's our time!
	// Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//Read P from the device    
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);        
	// Free device matrices    
	hipFree(Md); hipFree(Nd); hipFree (Pd); 
}

inline void MatrixMulOnDevice3(float* M, float* N, float* P, int Width, float timing) {   
	int size = Width * Width * sizeof(float);    
	float *Md, *Nd, *Pd;   
	//Allocate and Load M, N to device memory    
	hipMalloc(&Md, size);   
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);   
	hipMalloc(&Nd, size);   
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);   
	//Allocate P on the device   
	hipMalloc(&Pd, size);
	// Kernel invocation code � to be shown later    
	// Setup the execution configuration    
	dim3 dimGrid3(Width/16, Width/16);   
	dim3 dimBlock3(16, 16);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// Start record
	hipEventRecord(start, 0);
	// Do something on GPU
    // Launch the device computation threads!  
    // Launch the device computation threads!    
	MatrixMulKernelTiled16x16<<< dimGrid3, dimBlock3 >>>(Md, Nd, Pd, Width);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timing, start, stop); // that's our time!
	// Clean up:
	hipEventDestroy(start);
	hipEventDestroy(stop);
	//Read P from the device    
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);        
	// Free device matrices    
	hipFree(Md); hipFree(Nd); hipFree (Pd); 
}

inline void MatrixMulOnDevice4(float* M, float* N, float* P, int Width, int unrolling) {   
	int size = Width * Width * sizeof(float);    
	float *Md, *Nd, *Pd;   
	//Allocate and Load M, N to device memory    
	hipMalloc(&Md, size);   
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);   
	hipMalloc(&Nd, size);   
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);   
	//Allocate P on the device   
	hipMalloc(&Pd, size);
	// Kernel invocation code � to be shown later    
	// Setup the execution configuration    
	dim3 dimGrid3(Width/8, Width/8);   
	dim3 dimBlock3(8, 8);

    // Launch the device computation threads!    
	MatrixMulKernelTiledUnrolling<<< dimGrid3, dimBlock3 >>>(Md, Nd, Pd, Width, unrolling);

	//Read P from the device    
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);        
	// Free device matrices    
	hipFree(Md); hipFree(Nd); hipFree (Pd); 
}

inline void MatrixMulOnDevice5(float* M, float* N, float* P, int Width) {   
	int size = Width * Width * sizeof(float);    
	float *Md, *Nd, *Pd;   
	//Allocate and Load M, N to device memory    
	hipMalloc(&Md, size);   
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);   
	hipMalloc(&Nd, size);   
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);   
	//Allocate P on the device   
	hipMalloc(&Pd, size);
	// Kernel invocation code � to be shown later    
	// Setup the execution configuration    
	dim3 dimGrid3(Width/8, Width/8);   
	dim3 dimBlock3(8, 8);

    // Launch the device computation threads!    
	MatrixMulKernelTiled8x8prefetch<<< dimGrid3, dimBlock3 >>>(Md, Nd, Pd, Width);

	//Read P from the device    
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);        
	// Free device matrices    
	hipFree(Md); hipFree(Nd); hipFree (Pd); 
}

inline void MatrixMulOnDevice6(float* M, float* N, float* P, int Width, int gran) {   
	int size = Width * Width * sizeof(float);    
	float *Md, *Nd, *Pd;   
	//Allocate and Load M, N to device memory    
	hipMalloc(&Md, size);   
	hipMemcpy(Md, M, size, hipMemcpyHostToDevice);   
	hipMalloc(&Nd, size);   
	hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);   
	//Allocate P on the device   
	hipMalloc(&Pd, size);
	// Kernel invocation code � to be shown later    
	// Setup the execution configuration    
	dim3 dimGrid3(Width/8, Width/8);   
	dim3 dimBlock3(8, 8);

    // Launch the device computation threads! 
	if(gran == 2) {
		MatrixMulKernelTiled8x8gran1x2<<< dimGrid3, dimBlock3 >>>(Md, Nd, Pd, Width);
	}
	else if(gran == 4) {
		MatrixMulKernelTiled8x8gran1x4<<< dimGrid3, dimBlock3 >>>(Md, Nd, Pd, Width);
	}

	//Read P from the device    
	hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);        
	// Free device matrices    
	hipFree(Md); hipFree(Nd); hipFree (Pd); 
}


// Matrix multiplication kernel � per thread code
/*inline __global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width) {        
	// Pvalue is used to store the element of the matrix    
	// that is computed by the thread    
	float Pvalue = 0;
	for (int k = 0; k < Width; ++k) {    
		float Melement=Md[threadIdx.y*Width+k];    
		float Nelement=Nd[k*Width+threadIdx.x];    
		Pvalue += Melement * Nelement;  
	}  
	Pd[threadIdx.y*Width+threadIdx.x]=Pvalue; 
}*/


inline __global__ void MatrixMulKernelTiled8x8(float* Md, float* Nd, float* Pd, int Width) { 
	const int TILE_WIDTH = 8;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
		Nds[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k) 
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled16x16(float* Md, float* Nd, float* Pd, int Width) { 
	const int TILE_WIDTH = 16;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
		Nds[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k) 
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled8x8Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	const int TILE_WIDTH = 8;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
		Nds[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();

		if(unrolling == 2) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
			}
		}
		else if(unrolling == 4) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
				Pvalue += Mds[ty][k + 2] * Nds[k + 2][tx];
				Pvalue += Mds[ty][k + 3] * Nds[k + 3][tx];
			}
		}
		else {
			Pvalue += Mds[ty][0] * Nds[0][tx];
			Pvalue += Mds[ty][1] * Nds[1][tx];
			Pvalue += Mds[ty][2] * Nds[2][tx];
			Pvalue += Mds[ty][3] * Nds[3][tx];
			Pvalue += Mds[ty][4] * Nds[4][tx];
			Pvalue += Mds[ty][5] * Nds[5][tx];
			Pvalue += Mds[ty][6] * Nds[6][tx];
			Pvalue += Mds[ty][7] * Nds[7][tx];
		}
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled16x16Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	const int TILE_WIDTH = 16;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
		Nds[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col];
		__syncthreads();

		if(unrolling == 2) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
			}
		}
		else if(unrolling == 4) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
				Pvalue += Mds[ty][k + 2] * Nds[k + 2][tx];
				Pvalue += Mds[ty][k + 3] * Nds[k + 3][tx];
			}
		}
		else {
			Pvalue += Mds[ty][0] * Nds[0][tx];
			Pvalue += Mds[ty][1] * Nds[1][tx];
			Pvalue += Mds[ty][2] * Nds[2][tx];
			Pvalue += Mds[ty][3] * Nds[3][tx];
			Pvalue += Mds[ty][4] * Nds[4][tx];
			Pvalue += Mds[ty][5] * Nds[5][tx];
			Pvalue += Mds[ty][6] * Nds[6][tx];
			Pvalue += Mds[ty][7] * Nds[7][tx];
		}
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled8x8prefetchUnrolling(float* Md, float* Nd, float* Pd, int unrolling) { 
	const int TILE_WIDTH = 8;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 

	int M = Md[Row*Width + (0*TILE_WIDTH + tx)]; 
	int N = Nd[(0*TILE_WIDTH + ty)*Width + Col];
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		Mds[ty][tx] = M;
		Nds[ty][tx] = N;
		__syncthreads();

		M = Md[Row*Width + ((m+1)*TILE_WIDTH + tx)]; 
		N = Nd[((m+1)*TILE_WIDTH + ty)*Width + Col];
		
		if(unrolling == 2) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
			}
		}
		else if(unrolling == 4) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
				Pvalue += Mds[ty][k + 2] * Nds[k + 2][tx];
				Pvalue += Mds[ty][k + 3] * Nds[k + 3][tx];
			}
		}
		else {
			Pvalue += Mds[ty][0] * Nds[0][tx];
			Pvalue += Mds[ty][1] * Nds[1][tx];
			Pvalue += Mds[ty][2] * Nds[2][tx];
			Pvalue += Mds[ty][3] * Nds[3][tx];
			Pvalue += Mds[ty][4] * Nds[4][tx];
			Pvalue += Mds[ty][5] * Nds[5][tx];
			Pvalue += Mds[ty][6] * Nds[6][tx];
			Pvalue += Mds[ty][7] * Nds[7][tx];
		}
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled16x16prefetchUnrolling(float* Md, float* Nd, float* Pd, int unrolling) { 
	const int TILE_WIDTH = 16;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 

	int M = Md[Row*Width + (0*TILE_WIDTH + tx)]; 
	int N = Nd[(0*TILE_WIDTH + ty)*Width + Col];
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		Mds[ty][tx] = M;
		Nds[ty][tx] = N;
		__syncthreads();

		M = Md[Row*Width + ((m+1)*TILE_WIDTH + tx)]; 
		N = Nd[((m+1)*TILE_WIDTH + ty)*Width + Col];
		
		if(unrolling == 2) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
			}
		}
		else if(unrolling == 4) {
			for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
				Pvalue += Mds[ty][k] * Nds[k][tx];
				Pvalue += Mds[ty][k + 1] * Nds[k + 1][tx];
				Pvalue += Mds[ty][k + 2] * Nds[k + 2][tx];
				Pvalue += Mds[ty][k + 3] * Nds[k + 3][tx];
			}
		}
		else {
			Pvalue += Mds[ty][0] * Nds[0][tx];
			Pvalue += Mds[ty][1] * Nds[1][tx];
			Pvalue += Mds[ty][2] * Nds[2][tx];
			Pvalue += Mds[ty][3] * Nds[3][tx];
			Pvalue += Mds[ty][4] * Nds[4][tx];
			Pvalue += Mds[ty][5] * Nds[5][tx];
			Pvalue += Mds[ty][6] * Nds[6][tx];
			Pvalue += Mds[ty][7] * Nds[7][tx];
		}
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled8x8prefetch(float* Md, float* Nd, float* Pd, int Width) { 
	const int TILE_WIDTH = 8;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 

	int M = Md[Row*Width + (0*TILE_WIDTH + tx)]; 
	int N = Nd[(0*TILE_WIDTH + ty)*Width + Col];
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		Mds[ty][tx] = M;
		Nds[ty][tx] = N;
		__syncthreads();

		M = Md[Row*Width + ((m+1)*TILE_WIDTH + tx)]; 
		N = Nd[((m+1)*TILE_WIDTH + ty)*Width + Col];
		
		for (int k = 0; k < TILE_WIDTH; ++k) 
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled16x16prefetch(float* Md, float* Nd, float* Pd, int Width) { 
	const int TILE_WIDTH = 16;
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH]; 
	int bx = blockIdx.x;  
	int by = blockIdx.y; 
	int tx = threadIdx.x; 
	int ty = threadIdx.y; 
	// Identify the row and column of the Pd element to work on
	int Row = by * TILE_WIDTH + ty; 
	int Col = bx * TILE_WIDTH + tx; 
	float Pvalue = 0; 
	// Loop over the Md and Nd tiles required to compute the Pd element 

	int M = Md[Row*Width + (0*TILE_WIDTH + tx)]; 
	int N = Nd[(0*TILE_WIDTH + ty)*Width + Col];
	for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
		Mds[ty][tx] = M;
		Nds[ty][tx] = N;
		__syncthreads();

		M = Md[Row*Width + ((m+1)*TILE_WIDTH + tx)]; 
		N = Nd[((m+1)*TILE_WIDTH + ty)*Width + Col];
		
		for (int k = 0; k < TILE_WIDTH; ++k) 
			Pvalue += Mds[ty][k] * Nds[k][tx];
		__syncthreads();
	} 
	Pd[Row*Width + Col] = Pvalue; 
}

inline __global__ void MatrixMulKernelTiled8x8gran1x2(float* Md, float* Nd, float* Pd, int Width) { 
	if ((blockIdx.x % 2) == 0) {	
		const int TILE_WIDTH = 8;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
	}
}

inline __global__ void MatrixMulKernelTiled8x8gran1x4(float* Md, float* Nd, float* Pd, int Width) { 
	if ((blockIdx.x % 4) == 0) {
		const int TILE_WIDTH = 8;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds3[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds4[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		int Col3 = Col2 + TILE_WIDTH; 
		int Col4 = Col3 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		float Pvalue3 = 0; 
		float Pvalue4 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			Nds3[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col3];
			Nds4[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col4];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
		Pd[Row*Width + Col3] = Pvalue3; 
		Pd[Row*Width + Col4] = Pvalue4; 
	}
}

inline __global__ void MatrixMulKernelTiled16x16gran1x2(float* Md, float* Nd, float* Pd, int Width) { 
	if ((blockIdx.x % 2) == 0) {	
		const int TILE_WIDTH = 16;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
	}
}

inline __global__ void MatrixMulKernelTiled16x16gran1x4(float* Md, float* Nd, float* Pd, int Width) { 
	if ((blockIdx.x % 4) == 0) {
		const int TILE_WIDTH = 16;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds3[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds4[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		int Col3 = Col2 + TILE_WIDTH; 
		int Col4 = Col3 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		float Pvalue3 = 0; 
		float Pvalue4 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			Nds3[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col3];
			Nds4[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col4];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
		Pd[Row*Width + Col3] = Pvalue3; 
		Pd[Row*Width + Col4] = Pvalue4; 
	}
}

inline __global__ void MatrixMulKernelTiled8x8gran1x2Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	if ((blockIdx.x % 2) == 0) {	
		const int TILE_WIDTH = 8;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			__syncthreads();
			if(unrolling == 2) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				}
			}
			else if(unrolling == 4) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
					Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
					Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
					Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				}
			}
			else {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
				Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
				Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
				Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
				Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				Pvalue1 += Mds[ty][k+4] * Nds1[k+4][tx];
				Pvalue2 += Mds[ty][k+4] * Nds2[k+4][tx];
				Pvalue1 += Mds[ty][k+5] * Nds1[k+5][tx];
				Pvalue2 += Mds[ty][k+5] * Nds2[k+5][tx];
				Pvalue1 += Mds[ty][k+6] * Nds1[k+6][tx];
				Pvalue2 += Mds[ty][k+6] * Nds2[k+6][tx];
				Pvalue1 += Mds[ty][k+7] * Nds1[k+7][tx];
				Pvalue2 += Mds[ty][k+7] * Nds2[k+7][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
	}
}

inline __global__ void MatrixMulKernelTiled8x8gran1x4Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	if ((blockIdx.x % 4) == 0) {
		const int TILE_WIDTH = 8;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds3[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds4[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		int Col3 = Col2 + TILE_WIDTH; 
		int Col4 = Col3 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		float Pvalue3 = 0; 
		float Pvalue4 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			Nds3[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col3];
			Nds4[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col4];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
			}
			if(unrolling == 2) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue3 += Mds[ty][k] * Nds3[k][tx];
					Pvalue4 += Mds[ty][k] * Nds4[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
					Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
				}
			}
			else if(unrolling == 4) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue3 += Mds[ty][k] * Nds3[k][tx];
					Pvalue4 += Mds[ty][k] * Nds4[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
					Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
					Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
					Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
					Pvalue3 += Mds[ty][k+2] * Nds3[k+2][tx];
					Pvalue4 += Mds[ty][k+2] * Nds4[k+2][tx];
					Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
					Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
					Pvalue3 += Mds[ty][k+3] * Nds3[k+3][tx];
					Pvalue4 += Mds[ty][k+3] * Nds4[k+3][tx];
				}
			}
			else {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
				Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
				Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
				Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
				Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
				Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
				Pvalue3 += Mds[ty][k+2] * Nds3[k+2][tx];
				Pvalue4 += Mds[ty][k+2] * Nds4[k+2][tx];
				Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
				Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				Pvalue3 += Mds[ty][k+3] * Nds3[k+3][tx];
				Pvalue4 += Mds[ty][k+3] * Nds4[k+3][tx];
				Pvalue1 += Mds[ty][k+4] * Nds1[k+4][tx];
				Pvalue2 += Mds[ty][k+4] * Nds2[k+4][tx];
				Pvalue3 += Mds[ty][k+4] * Nds3[k+4][tx];
				Pvalue4 += Mds[ty][k+4] * Nds4[k+4][tx];
				Pvalue1 += Mds[ty][k+5] * Nds1[k+5][tx];
				Pvalue2 += Mds[ty][k+5] * Nds2[k+5][tx];
				Pvalue3 += Mds[ty][k+5] * Nds3[k+5][tx];
				Pvalue4 += Mds[ty][k+5] * Nds4[k+5][tx];
				Pvalue1 += Mds[ty][k+6] * Nds1[k+6][tx];
				Pvalue2 += Mds[ty][k+6] * Nds2[k+6][tx];
				Pvalue3 += Mds[ty][k+6] * Nds3[k+6][tx];
				Pvalue4 += Mds[ty][k+6] * Nds4[k+6][tx];
				Pvalue1 += Mds[ty][k+7] * Nds1[k+7][tx];
				Pvalue2 += Mds[ty][k+7] * Nds2[k+7][tx];
				Pvalue3 += Mds[ty][k+7] * Nds3[k+7][tx];
				Pvalue4 += Mds[ty][k+7] * Nds4[k+7][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
		Pd[Row*Width + Col3] = Pvalue3; 
		Pd[Row*Width + Col4] = Pvalue4; 
	}
}

inline __global__ void MatrixMulKernelTiled16x16gran1x2Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	if ((blockIdx.x % 2) == 0) {	
		const int TILE_WIDTH = 16;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			__syncthreads();
			if(unrolling == 2) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				}
			}
			else if(unrolling == 4) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
					Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
					Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
					Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				}
			}
			else {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
				Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
				Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
				Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
				Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				Pvalue1 += Mds[ty][k+4] * Nds1[k+4][tx];
				Pvalue2 += Mds[ty][k+4] * Nds2[k+4][tx];
				Pvalue1 += Mds[ty][k+5] * Nds1[k+5][tx];
				Pvalue2 += Mds[ty][k+5] * Nds2[k+5][tx];
				Pvalue1 += Mds[ty][k+6] * Nds1[k+6][tx];
				Pvalue2 += Mds[ty][k+6] * Nds2[k+6][tx];
				Pvalue1 += Mds[ty][k+7] * Nds1[k+7][tx];
				Pvalue2 += Mds[ty][k+7] * Nds2[k+7][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
	}
}

inline __global__ void MatrixMulKernelTiled16x16gran1x4Unrolling(float* Md, float* Nd, float* Pd, int Width, int unrolling) { 
	if ((blockIdx.x % 4) == 0) {
		const int TILE_WIDTH = 16;
		__shared__ float Mds[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds1[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds2[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds3[TILE_WIDTH][TILE_WIDTH]; 
		__shared__ float Nds4[TILE_WIDTH][TILE_WIDTH]; 
		int bx = blockIdx.x;  
		int by = blockIdx.y; 
		int tx = threadIdx.x; 
		int ty = threadIdx.y; 
		// Identify the row and column of the Pd element to work on
		int Row = by * TILE_WIDTH + ty; 
		int Col1 = bx * TILE_WIDTH + tx; 
		int Col2 = Col1 + TILE_WIDTH; 
		int Col3 = Col2 + TILE_WIDTH; 
		int Col4 = Col3 + TILE_WIDTH; 
		float Pvalue1 = 0; 
		float Pvalue2 = 0; 
		float Pvalue3 = 0; 
		float Pvalue4 = 0; 
		// Loop over the Md and Nd tiles required to compute the Pd element 
		for (int m = 0; m < Width/TILE_WIDTH; ++m) { 
			// Collaborative loading of Md and Nd tiles into shared memory
			Mds[ty][tx] = Md[Row*Width + (m*TILE_WIDTH + tx)]; 
			Nds1[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col1];
			Nds2[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col2];
			Nds3[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col3];
			Nds4[ty][tx] = Nd[(m*TILE_WIDTH + ty)*Width + Col4];
			__syncthreads();
			for (int k = 0; k < TILE_WIDTH; ++k) {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
			}
			if(unrolling == 2) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue3 += Mds[ty][k] * Nds3[k][tx];
					Pvalue4 += Mds[ty][k] * Nds4[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
					Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
				}
			}
			else if(unrolling == 4) {
				for (int k = 0; k < TILE_WIDTH; k+=unrolling) {
					Pvalue1 += Mds[ty][k] * Nds1[k][tx];
					Pvalue2 += Mds[ty][k] * Nds2[k][tx];
					Pvalue3 += Mds[ty][k] * Nds3[k][tx];
					Pvalue4 += Mds[ty][k] * Nds4[k][tx];
					Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
					Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
					Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
					Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
					Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
					Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
					Pvalue3 += Mds[ty][k+2] * Nds3[k+2][tx];
					Pvalue4 += Mds[ty][k+2] * Nds4[k+2][tx];
					Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
					Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
					Pvalue3 += Mds[ty][k+3] * Nds3[k+3][tx];
					Pvalue4 += Mds[ty][k+3] * Nds4[k+3][tx];
				}
			}
			else {
				Pvalue1 += Mds[ty][k] * Nds1[k][tx];
				Pvalue2 += Mds[ty][k] * Nds2[k][tx];
				Pvalue3 += Mds[ty][k] * Nds3[k][tx];
				Pvalue4 += Mds[ty][k] * Nds4[k][tx];
				Pvalue1 += Mds[ty][k+1] * Nds1[k+1][tx];
				Pvalue2 += Mds[ty][k+1] * Nds2[k+1][tx];
				Pvalue3 += Mds[ty][k+1] * Nds3[k+1][tx];
				Pvalue4 += Mds[ty][k+1] * Nds4[k+1][tx];
				Pvalue1 += Mds[ty][k+2] * Nds1[k+2][tx];
				Pvalue2 += Mds[ty][k+2] * Nds2[k+2][tx];
				Pvalue3 += Mds[ty][k+2] * Nds3[k+2][tx];
				Pvalue4 += Mds[ty][k+2] * Nds4[k+2][tx];
				Pvalue1 += Mds[ty][k+3] * Nds1[k+3][tx];
				Pvalue2 += Mds[ty][k+3] * Nds2[k+3][tx];
				Pvalue3 += Mds[ty][k+3] * Nds3[k+3][tx];
				Pvalue4 += Mds[ty][k+3] * Nds4[k+3][tx];
				Pvalue1 += Mds[ty][k+4] * Nds1[k+4][tx];
				Pvalue2 += Mds[ty][k+4] * Nds2[k+4][tx];
				Pvalue3 += Mds[ty][k+4] * Nds3[k+4][tx];
				Pvalue4 += Mds[ty][k+4] * Nds4[k+4][tx];
				Pvalue1 += Mds[ty][k+5] * Nds1[k+5][tx];
				Pvalue2 += Mds[ty][k+5] * Nds2[k+5][tx];
				Pvalue3 += Mds[ty][k+5] * Nds3[k+5][tx];
				Pvalue4 += Mds[ty][k+5] * Nds4[k+5][tx];
				Pvalue1 += Mds[ty][k+6] * Nds1[k+6][tx];
				Pvalue2 += Mds[ty][k+6] * Nds2[k+6][tx];
				Pvalue3 += Mds[ty][k+6] * Nds3[k+6][tx];
				Pvalue4 += Mds[ty][k+6] * Nds4[k+6][tx];
				Pvalue1 += Mds[ty][k+7] * Nds1[k+7][tx];
				Pvalue2 += Mds[ty][k+7] * Nds2[k+7][tx];
				Pvalue3 += Mds[ty][k+7] * Nds3[k+7][tx];
				Pvalue4 += Mds[ty][k+7] * Nds4[k+7][tx];
			}
			__syncthreads();
		} 
		Pd[Row*Width + Col1] = Pvalue1; 
		Pd[Row*Width + Col2] = Pvalue2; 
		Pd[Row*Width + Col3] = Pvalue3; 
		Pd[Row*Width + Col4] = Pvalue4; 
	}
}