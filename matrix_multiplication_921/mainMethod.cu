
#include "hip/hip_runtime.h"
#include ""
#include "matrixMultiply.cu"
#include "matrixGenerator.cu"

#include <stdio.h>

int main()
{

    const int arraySize = 512;
    float a[arraySize*arraySize];
	for(int i = 0; i < 262144; i++) {
		a[i] = 2;
	}
    float b[arraySize*arraySize];
	for(int i = 0; i < 262144; i++) {
		b[i] = 2;
	}
    float c[arraySize*arraySize] = { 0 };
	float d[arraySize*arraySize] = { 0 };
	float e[arraySize*arraySize] = { 0 };
	float f[arraySize*arraySize] = { 0 };
	float timing2 = 0;
	float timing3 = 0;

	MatrixMulOnDevice2(a, b, c, arraySize, timing2);
	MatrixMulOnDevice3(a, b, d, arraySize, timing3);
	MatrixMulOnDevice4(a, b, e, arraySize, 2);
	MatrixMulOnDevice4(a, b, f, arraySize, 4);

	printf("{%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n}",
        c[0], c[1], c[2], c[3], c[4], c[5], c[6], c[7], c[8], c[9], c[10], c[11], c[12], c[13], c[14], c[15]);
	printf("The time is: %f", timing2);
	printf("\n");
	printf("{%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n}",
        d[0], d[1], d[2], d[3], d[4], d[5], d[6], d[7], d[8], d[9], d[10], d[11], d[12], d[13], d[14], d[15]);
	printf("The time is: %f", timing3);
	printf("\n");
	printf("{%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n}",
        e[0], e[1], e[2], e[3], e[4], e[5], e[6], e[7], e[8], e[9], e[10], e[11], e[12], e[13], e[14], e[15]);
	printf("{%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n,%f\n}",
        f[0], f[1], f[2], f[3], f[4], f[5], f[6], f[7], f[8], f[9], f[10], f[11], f[12], f[13], f[14], f[15]);
	getchar();

}
